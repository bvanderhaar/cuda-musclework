#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

/*
 * In CUDA it is necessary to define block sizes
 * The grid of data that will be worked on is divided into blocks
 */
#define BLOCK_SIZE 8
/**
 * The function that will be executed in each stream processors
 * The __global__ directive identifies this function as being
 * an executable kernel on the CUDA device.
 * All kernesl must be declared with a return type void
 */
__global__ void cu_dotProduct(int *block_d, int *thread_d) {
  int x;
  /* blockIdx.x is a built-in variable in CUDA
     that returns the blockId in the x axis.
     threadIdx.x is another built-in variable in CUDA
     that returns the threadId in the x axis
     of the thread that is being executed by the
     stream processor accessing this particular block
  */
  x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  block_d[x] = blockIdx.x;
  thread_d[x] = threadIdx.x;
}

// Called from driver program.  Handles running GPU calculation
extern "C" void gpu_dotProduct(int *distance_array, int *force_array,
                               int num_vectors) {
  // a_d is the GPU counterpart of the array that exists in host memory
  int *distance_array_d;
  int *force_array_d;
  int result_array[num_vertices];
  int *result_array_d;
  hipError_t result;

  // allocate space in the device
  result = hipMalloc((void **)&distance_array_d, sizeof(int) * num_vertices);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMalloc failed.");
    exit(1);
  }

  result = hipMalloc((void **)&force_array_d, sizeof(int) * num_vertices);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMalloc failed.");
    exit(1);
  }

  result = hipMalloc((void **)&result_array_d, sizeof(int) * num_vertices);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMalloc failed.");
    exit(1);
  }

  // copy the array from host to array_d in the device
  result = hipMemcpy(distance_array_d, distance_array,
                      sizeof(int) * num_vertices, hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed.");
    exit(1);
  }

  result = hipMemcpy(force_array_d, force_array, sizeof(int) * num_vertices,
                      hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed.");
    exit(1);
  }

  result = hipMemcpy(result_array_d, result_array, sizeof(int) * num_vertices,
                      hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed.");
    exit(1);
  }

  // set execution configuration
  dim3 dimblock(BLOCK_SIZE);
  dim3 dimgrid(num_vertices / BLOCK_SIZE);

  // actual computation: Call the kernel
  cu_dotProduct<<<dimgrid, dimblock>>>(result_array_d);

  // transfer results back to host
  result = hipMemcpy(result_array, result_array_d, sizeof(int) * arraySize,
                      hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed.");
    exit(1);
  }

  // release the memory on the GPU
  result = hipFree(distance_array_d);
  if (result != hipSuccess) {
    fprintf(stderr, "hipFree failed.");
    exit(1);
  }

  // release the memory on the GPU
  result = hipFree(force_array_d);
  if (result != hipSuccess) {
    fprintf(stderr, "hipFree failed.");
    exit(1);
  }

  // release the memory on the GPU
  result = hipFree(result_array_d);
  if (result != hipSuccess) {
    fprintf(stderr, "hipFree failed.");
    exit(1);
  }
}
