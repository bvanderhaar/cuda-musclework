
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
 * In CUDA it is necessary to define block sizes
 * The grid of data that will be worked on is divided into blocks
 */
#define BLOCK_SIZE 512

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void cu_dotProduct(long long *distance_array_d,
                              long long *force_array_d,
                              long long *result_array_d, long long max) {
  long long x;
  x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  if (x < max) {
    result_array_d[x] = distance_array_d[x] * force_array_d[x];
  }
}

__global__ void cu_gen_force_array(long long *force_array_d, long long max) {
  long long x, half_vectors;
  x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  half_vectors = max / 2;
  if (x < half_vectors) {
    force_array_d[x] = x + 1;
  } else {
    force_array_d[x] = half_vectors + (half_vectors - x);
  }
}

__global__ void cu_gen_distance_array(long long *distance_array_d,
                                      long long max) {
  long long x;
  x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  distance_array_d[x] = (x + 1) % 10;
  if (distance_array_d[x] == 0) {
    distance_array_d[x] = 10;
  }
}

// Called from driver program.  Handles running GPU calculation
extern "C" void gpu_dotProduct(long long *result_array, long long num_vectors) {
  long long *distance_array_d;
  long long *force_array_d;
  long long *result_array_d;

  // allocate space in the device
  hipMalloc((void **)&distance_array_d, sizeof(long long) * num_vectors);
  hipMalloc((void **)&force_array_d, sizeof(long long) * num_vectors);
  hipMalloc((void **)&result_array_d, sizeof(long long) * num_vectors);

  // set execution configuration
  dim3 dimblock(BLOCK_SIZE);
  dim3 dimgrid(ceil((long double)num_vectors / BLOCK_SIZE));

  cu_gen_force_array<<<dimgrid, dimblock>>>(force_array_d, num_vectors);
  cu_gen_distance_array<<<dimgrid, dimblock>>>(distance_array_d, num_vectors);
  cu_dotProduct<<<dimgrid, dimblock>>>(distance_array_d, force_array_d,
                                       result_array_d, num_vectors);
  // transfer results back to host
  hipMemcpy(result_array, result_array_d, sizeof(long long) * num_vectors,
             hipMemcpyDeviceToHost);

  // release the memory on the GPU
  hipFree(distance_array_d);
  hipFree(force_array_d);
  hipFree(result_array_d);
}
